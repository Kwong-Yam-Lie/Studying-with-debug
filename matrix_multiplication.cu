
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include<stdlib.h>
//#include<windows.h>
//#include<cmath>
//#include<iostream>
//#include<omp.h>
//using namespace std;
//#define num_thread 29 // ��block�߳��� // ע��һ����ֵ�(num_thread,num)->(30,1000)�ٶȱ�(32,1000)��10����
//#define num 3000 // �����С 
////�������ԣ��˷�����ѡȡnum_thread=29������ѣ�num=1000ʱ����Ч�� 1300����num=3000ʱ����Ч�� 7500��
////��ģ������һ���£�������һ���¡���Ƶ��ٺ�ҲҪ��������������������Ʋ���ȥ��������Ч��Ҳ���Ե��Ĳ�����֤�����ɡ�
//
///*shared memory ��ʹ����Ҫ��Ϊ�˼��ٶ�ȫ���ڴ���ظ����ʴ������Դ�Ϊ��Ʊ�׼*/
///*Ϊ�˼ӿ�����ٶ�����һ���־��ȣ�ʹ��float���͡�int�����Ż����ã�double�����ֽ����ӱ�����Ȼfloat���ͻ��������ʧ��������ƽ����������������Խ���*/
///*ȱ�㣺kernel����̫��if���*/
///*"���ȼ�"˼�����ڳ����bug��������ĺ�ʹ������*/
//
///*����ǧ�����գ����ղ����bug:
//* 1. ��blockDim����gridDimʹ�ˣ�
//* 2. ���˹����ֵ�ִ�п��ά�ȴ�С�ȼ���ʵ�ʳ����block��С�ˣ�������������ǰ�dev_C����Ϊ���ִ�п飬ÿ�����С(num_thread,num_thread)
//*	 ���ǣ�ʵ����һ���߳�ִ��һ��ά�ȣ��Ҷ����block��(num_thread,num_thread)������ʹ��blockDim.y��1������num_thread����ɻ�������
//*/
//
//__global__ void matrix_mul_kernel(float* dev_C, float* dev_A, float* dev_B, int size, int weight, int height)
//{
//	/* allocation shared memory */ //(����������ķ����汾��ʵ���Ϻܶ���ⲻ��λ)
//	
//	/*�豸���
//	* ʹ��GPU device 0: Tesla V100S-PCIE-32GB
//	* SM��������80
//	* ÿ���߳̿�Ĺ����ڴ��С��48 KB
//	* ÿ���߳̿������߳�����1024
//	* ÿ��SM������߳�����2048
//	* ÿ��SM������߳�������64
//	*/
//
//	/*����
//	* 1. ��������Ĭ��ʹ�þ�̬�ڴ�Ϊ16k�������򾯸棬���������ⲻ��debug����Ҫ�����Ƿ��Ҫ������
//	*    Ƭ���ڴ�48k��Ҫ��block������á���ÿ�� SM 3��block����1�Ļ��������3*80=240��block��
//	*    ����һ��blockִ������һ��block��������ȥ������block�������ÿ����ˣ������ڴ��ڴ�ʹ���Ͽ��ǡ�
//	*	 ������ǣ�ÿ��SM 3��block��ÿ��block 2048/3=682 -> 32*21=672��thread�����16k�Ĺ����ڴ�
//	*
//	* 2. ÿ��block�Ĺ����ڴ涼����������飬(pow(x,2)+2*x)*4.0/1024=16,���������x=63
//	*	 ��ôÿ��blockӦ��������63��thread����ô���Կ���1��2���໥��Լ�ġ�
//		 Ӧ������ѡȡ���ʵ�num_threads?Ӧ��ѡȡʲôά�ȵ�block��
//	*
//	* 3. ��ʩһ����block����16k���棬ֱ����������ʱ���������x=109 -> x=32*3=96������SM����ִ��һ��block��
//	*	 ��ʩ�����ѼĴ�����16k�ù���������������̫�ã����������ٶȣ�����Ҳ�ѿز�ס��
//	*	 ��ʩ����ѡȡ1ά��block����ά�Ļ�num_threadsҪ���������ܶ�ά��num_threads����Ƴ��򣬵������ֶ�ȫ�ֱ������ʵ��ظ����������
//	*
//	* 4. ���������Grid�϶�άblock;blockDim.y=96,blockDim.x=1;ÿ���̵߳���ִ��1�����ز���������weight�Ρ�
//	*/
//	constexpr int N{ num_thread };
//	__shared__ float temp_A[N * 1]; // 96��֮��������64
//	__shared__ float temp_B[1 * N];
//	__shared__ float temp_C[N][N]; // [size_y][size_x] ��ά����������ģ�
//
//	for (int i = 0; i < N; ++i) { // �Ȱ�temp_C��ʼ����������á�+=�������
//		temp_C[threadIdx.x][i] = 0;
//	}
//	__syncthreads();
//
//	/*compute in each thread*/
//	if (blockIdx.x < (gridDim.x - 1)) {// �б߽�����������������
//		for (int count = 0; count < weight; ++count) {
//			temp_B[threadIdx.x] = dev_B[count * weight + (blockDim.x * blockIdx.x + threadIdx.x)];
//			if ((blockIdx.y * num_thread + threadIdx.x) < height) {
//				temp_A[threadIdx.x] = dev_A[(num_thread * blockIdx.y + threadIdx.x) * weight + count];
//			}// Ҫ��������㸳ֵ����Ҫ��������ݴ���ȥ����
//			else temp_A[threadIdx.x] = 10;
//			__syncthreads();
//
//			for (int col = 0; col < N; ++col) {
//				temp_C[threadIdx.x][col] += temp_A[threadIdx.x] * temp_B[col];
//			}
//			__syncthreads();
//		}
//		__syncthreads();
//		if ((blockIdx.y * num_thread + threadIdx.x) < height) {
//			for (int col = 0; col < N; ++col) {
//				dev_C[(blockIdx.y * num_thread + threadIdx.x) * weight + (blockIdx.x * blockDim.x + col)] = temp_C[threadIdx.x][col];
//			}// ���Ｔʹ����if�ж������ſ��Ҹ�ֵ��Ӹ�1����(num_thread,num)->(3,6)ʱ�������Ҳ���������
//		}	 // �̻߳������ִ�У����ǲ�������ô�ࡢ�Գ����������ô�࣬��������������û���ɰ������ǹ���
//	}
//
//	if (blockIdx.x == (gridDim.x - 1)) {// ������
//		for (int count = 0; count < weight; ++count) {
//			temp_A[threadIdx.x] = dev_A[(num_thread * blockIdx.y + threadIdx.x) * weight + count];
//			if ((blockIdx.x * blockDim.x + threadIdx.x) < weight) {
//				temp_B[threadIdx.x] = dev_B[count * weight + (blockDim.x * blockIdx.x + threadIdx.x)];
//			}
//			else temp_B[threadIdx.x] = 0;
//			__syncthreads();
//
//			for (int col = 0; col < N; ++col) {
//				temp_C[threadIdx.x][col] += temp_A[threadIdx.x] * temp_B[col];
//			}
//			__syncthreads();
//		}
//		__syncthreads();
//		for (int col = 0; col < (weight - blockIdx.x * blockDim.x); ++col) {
//			dev_C[(blockIdx.y * num_thread + threadIdx.x) * weight + (blockIdx.x * blockDim.x + col)] = temp_C[threadIdx.x][col];
//		}
//
//	}
//}
//
//void matrix_mul_withCuda(float* C, float* A, float* B, int size, int weight, int height)
//{
//	/*Device configuration*/
//	cudaSetDevice(0);
//
//	float* dev_C; float* dev_A; float* dev_B;
//	cudaMalloc((void**)&dev_C, size * sizeof(float));
//	cudaMalloc((void**)&dev_A, size * sizeof(float));
//	cudaMalloc((void**)&dev_B, size * sizeof(float));
//
//	cudaMemset(dev_C, 0, size * sizeof(float));
//	cudaMemcpy(dev_A, A, size * sizeof(float), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_B, B, size * sizeof(float), cudaMemcpyHostToDevice);
//
//	/*kernel : configuration*/
//	dim3 threadsPerBlock(num_thread); // �зֳɶ���block��ÿ��block��thread����һ��
//	// threadsPerblock.xԽ���ظ�����ȫ���ڴ����Խ�٣�����������ô��ã�������Խ��Խ�ã�������kernel
//	dim3 numBlocks((int)ceil((1.0 * height) / threadsPerBlock.x), (int)ceil((1.0 * height) / threadsPerBlock.x));
//	cout << "Block: " << (int)ceil((1.0 * height) / threadsPerBlock.x) << "," << (int)ceil((1.0 * height) / threadsPerBlock.x) << endl;
//	// ����Ŀ�����dev_C������block��ÿ��blockֻ��Ҫ96���̼߳������
//
//	//ʹ��event����ʱ��
//	float time_elapsed{ 0 };
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start); //����Event
//	cudaEventCreate(&stop);
//
//	cudaEventRecord(start, 0); //��¼��ǰʱ��
//	/*kernek : calling function*/
//	matrix_mul_kernel << <numBlocks, threadsPerBlock >> > (dev_C, dev_A, dev_B, size, weight, height);
//	cudaDeviceSynchronize();
//	cudaError_t err = cudaGetLastError();
//	if (err != cudaSuccess) {
//		printf("CUDA ERROR: %s\n", cudaGetErrorString(err));
//	}
//	cudaEventRecord(stop, 0); //��¼��ǰʱ��
//
//	cudaEventSynchronize(start); //Waits for an event to complete.
//	cudaEventSynchronize(stop);  //Waits for an event to complete.Record֮ǰ������
//	cudaEventElapsedTime(&time_elapsed, start, stop); //����ʱ���
//	cout << "CUDA����ʱ�䣨�������ݴ��䣩�� " << time_elapsed << "ms" << endl;
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	/*Device To Host*/
//	cudaMemcpy(C, dev_C, size * sizeof(float), cudaMemcpyDeviceToHost);
//
//	/*Free Memory*/
//	cudaFree(dev_A);	cudaFree(dev_B);	cudaFree(dev_C);
//
//}
//
//int main()
//{
//	constexpr int weight{ num }; constexpr int height{ weight };
//	constexpr int size{ weight * height };
//	float* A = new float[size] {};
//	float* B = new float[size] {};
//	float* C = new float[size] {};
//	srand((unsigned)time(NULL));
//	for (int i = 0; i < size; ++i) {
//		//A[i] = (rand() % 4) / ((float)1); // �������Ծ�������׼
//		//B[i] = (rand() % 4) / ((float)1);
//		A[i] = (rand() % 256) / ((float)255); // ��������ͨ������׼
//		B[i] = (rand() % 256) / ((float)255);
//	}
//	//cout << "*****A********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << A[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//	//cout << "*****B********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << B[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//	/*���о���˷�����*/
//	ULONGLONG t1, t2;
//	t1 = GetTickCount64();
//	float* CC = new float[size] {};
//	// nvcc ���� parallel for ��������
//	// Runtime���̣�2000*2000Ҫ75�룬1000*1000Ҫ2�룬���ݴ��˷�������
//	// CPP���̣�2000*2000Ҫ5�룬1000*1000Ҫ0.15��
////#pragma omp parallel for  
//	for (int i = 0; i < height; ++i) { // ����������
//		for (int j = 0; j < weight; ++j) {
//			for (int col = 0; col < weight; ++col) {
//				CC[i * weight + j] += A[i * weight + col] * B[col * weight + j];
//			}
//		}
//	}// 3000*3000�Ĵ�����Ҫʱ�� 318s
//	t2 = GetTickCount64();
//	cout << "���о���˷������ʱ�� " << t2 - t1 << "ms" << endl;
//
//	/*����CUDA���о���˷�����*/
//	matrix_mul_withCuda(C, A, B, size, weight, height);
//
//	double error{ 0 };
//	for (int i = 0; i < height; ++i) {
//		for (int j = 0; j < weight; ++j) {
//			error += abs(CC[i * weight + j] - C[i * weight + j]);
//		}
//	}
//	cout << "sum of absolute error: " << error << endl;
//	cout << "mean error: " << error / size << endl;
//	//cout << "*****CC********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << CC[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//	//cout << "*****C********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << C[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//
//	return 0;
//}
///*���Դ˳�������Լ�д�ĺܶ�����߼��Ͽ��Լ򻯴Ӷ����ٴ��������������������Ķ�Ҳ����debug*/
///*�߼������ݵĵ��Գ����ֶΣ� �й��ɸı丳ֵ�����ò����߼��ȣ��۲�Ԥ�ں�����Ƿ�һ��*/
///*�ҷ����ҵ�bug�����������߼�bug���������Ǵ���ֻ��߲�С��û���⼡�����˳дŪ�ɵĴ��󣡹ؼ�����Щ�����Լ������ϣ������øģ���*/