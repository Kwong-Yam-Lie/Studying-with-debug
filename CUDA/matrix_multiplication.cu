
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include<stdlib.h>
//#include<windows.h>
//#include<cmath>
//#include<iostream>
//#include<omp.h>
//using namespace std;
//#define num_thread 29 // 单block线程数 // 注意一个奇怪点(num_thread,num)->(30,1000)速度比(32,1000)快10倍数
//#define num 3000 // 方针大小 
////经过测试，此服务器选取num_thread=29性能最佳，num=1000时加速效果 1300倍；num=3000时加速效果 7500倍
////真的，设计是一回事，调参是一回事。设计的再好也要调调参数，反过来，设计差不多就去调参数，效果也可以调的不错，辩证看待吧。
//
///*shared memory 的使用主要是为了减少对全局内存的重复访问次数，以此为设计标准*/
///*为了加快计算速度牺牲一部分精度，使用float类型。int类型优化不好，double类型字节数加倍。虽然float类型会有误差损失，但是以平均误差来考量还可以接受*/
///*缺点：kernel函数太多if语句*/
///*"极度简化"思想用在程序调bug上是真真的好使！！！*/
//
///*历经千难万险，最终查出了bug:
//* 1. 把blockDim当成gridDim使了；
//* 2. 把人工划分的执行块的维度大小等价于实际程序的block大小了，比如这个程序是把dev_C划分为多个执行块，每个块大小(num_thread,num_thread)
//*	 但是，实际上一个线程执行一个维度，我定义的block是(num_thread,num_thread)，所以使用blockDim.y是1而不是num_thread，造成混淆错误
//*/
//
//__global__ void matrix_mul_kernel(float* dev_C, float* dev_A, float* dev_B, int size, int weight, int height)
//{
//	/* allocation shared memory */ //(下面是最初的分析版本，实际上很多理解不到位)
//	
//	/*设备情况
//	* 使用GPU device 0: Tesla V100S-PCIE-32GB
//	* SM的数量：80
//	* 每个线程块的共享内存大小：48 KB
//	* 每个线程块的最大线程数：1024
//	* 每个SM的最大线程数：2048
//	* 每个SM的最大线程束数：64
//	*/
//
//	/*分析
//	* 1. 单个函数默认使用静态内存为16k，超出则警告，后续有问题不好debug，需要衡量是否必要超出。
//	*    片上内存48k，要被block充分利用。若每个 SM 3个block，则按1的话单次最大3*80=240个block。
//	*    但是一个block执行完另一个block立即跟上去，所以block数量不用考虑了，着重于从内存使用上考虑。
//	*	 结果就是：每个SM 3个block，每个block 2048/3=682 -> 32*21=672个thread，最大16k的共享内存
//	*
//	* 2. 每个block的共享内存都有三个矩阵块，(pow(x,2)+2*x)*4.0/1024=16,解得整数解x=63
//	*	 那么每个block应该有至多63个thread，那么可以看出1和2是相互制约的。
//		 应该怎样选取合适的num_threads?应该选取什么维度的block？
//	*
//	* 3. 措施一：单block超出16k警告，直接拉满，这时解得整数解x=109 -> x=32*3=96，单个SM单次执行一个block。
//	*	 措施二：把寄存器的16k拿过来，但是这样不太好（可能拖慢速度），我也把控不住。
//	*	 措施三：选取1维的block，二维的话num_threads要开方，尽管二维的num_threads好设计程序，但这两种对全局变量访问的重复次数差多了
//	*
//	* 4. 分析结果：Grid上二维block;blockDim.y=96,blockDim.x=1;每个线程单次执行1个像素操作，迭代weight次。
//	*/
//	constexpr int N{ num_thread };
//	__shared__ float temp_A[N * 1]; // 96，之后再试验64
//	__shared__ float temp_B[1 * N];
//	__shared__ float temp_C[N][N]; // [size_y][size_x] 二维是这样定义的？
//
//	for (int i = 0; i < N; ++i) { // 先把temp_C初始化，后面好用“+=”运算符
//		temp_C[threadIdx.x][i] = 0;
//	}
//	__syncthreads();
//
//	/*compute in each thread*/
//	if (blockIdx.x < (gridDim.x - 1)) {// 列边界条件，不含最右列
//		for (int count = 0; count < weight; ++count) {
//			temp_B[threadIdx.x] = dev_B[count * weight + (blockDim.x * blockIdx.x + threadIdx.x)];
//			if ((blockIdx.y * num_thread + threadIdx.x) < height) {
//				temp_A[threadIdx.x] = dev_A[(num_thread * blockIdx.y + threadIdx.x) * weight + count];
//			}// 要不这里计算赋值错误，要不最后数据传回去错误
//			else temp_A[threadIdx.x] = 10;
//			__syncthreads();
//
//			for (int col = 0; col < N; ++col) {
//				temp_C[threadIdx.x][col] += temp_A[threadIdx.x] * temp_B[col];
//			}
//			__syncthreads();
//		}
//		__syncthreads();
//		if ((blockIdx.y * num_thread + threadIdx.x) < height) {
//			for (int col = 0; col < N; ++col) {
//				dev_C[(blockIdx.y * num_thread + threadIdx.x) * weight + (blockIdx.x * blockDim.x + col)] = temp_C[threadIdx.x][col];
//			}// 这里即使给它if判断条件放宽且赋值多加个1，在(num_thread,num)->(3,6)时最后两行也不会输出。
//		}	 // 线程还会挂起不执行？但是测试了这么多、对程序更改了这么多，但结果就是这个！没理由啊，真是怪了
//	}
//
//	if (blockIdx.x == (gridDim.x - 1)) {// 最右列
//		for (int count = 0; count < weight; ++count) {
//			temp_A[threadIdx.x] = dev_A[(num_thread * blockIdx.y + threadIdx.x) * weight + count];
//			if ((blockIdx.x * blockDim.x + threadIdx.x) < weight) {
//				temp_B[threadIdx.x] = dev_B[count * weight + (blockDim.x * blockIdx.x + threadIdx.x)];
//			}
//			else temp_B[threadIdx.x] = 0;
//			__syncthreads();
//
//			for (int col = 0; col < N; ++col) {
//				temp_C[threadIdx.x][col] += temp_A[threadIdx.x] * temp_B[col];
//			}
//			__syncthreads();
//		}
//		__syncthreads();
//		for (int col = 0; col < (weight - blockIdx.x * blockDim.x); ++col) {
//			dev_C[(blockIdx.y * num_thread + threadIdx.x) * weight + (blockIdx.x * blockDim.x + col)] = temp_C[threadIdx.x][col];
//		}
//
//	}
//}
//
//void matrix_mul_withCuda(float* C, float* A, float* B, int size, int weight, int height)
//{
//	/*Device configuration*/
//	cudaSetDevice(0);
//
//	float* dev_C; float* dev_A; float* dev_B;
//	cudaMalloc((void**)&dev_C, size * sizeof(float));
//	cudaMalloc((void**)&dev_A, size * sizeof(float));
//	cudaMalloc((void**)&dev_B, size * sizeof(float));
//
//	cudaMemset(dev_C, 0, size * sizeof(float));
//	cudaMemcpy(dev_A, A, size * sizeof(float), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_B, B, size * sizeof(float), cudaMemcpyHostToDevice);
//
//	/*kernel : configuration*/
//	dim3 threadsPerBlock(num_thread); // 行分成多少block，每个block的thread代表一行
//	// threadsPerblock.x越大重复访问全局内存次数越少，因此这里设置大点好，但不是越大越好，分析见kernel
//	dim3 numBlocks((int)ceil((1.0 * height) / threadsPerBlock.x), (int)ceil((1.0 * height) / threadsPerBlock.x));
//	cout << "Block: " << (int)ceil((1.0 * height) / threadsPerBlock.x) << "," << (int)ceil((1.0 * height) / threadsPerBlock.x) << endl;
//	// 根据目标矩阵dev_C来划分block，每个block只需要96个线程即可求解
//
//	//使用event计算时间
//	float time_elapsed{ 0 };
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start); //创建Event
//	cudaEventCreate(&stop);
//
//	cudaEventRecord(start, 0); //记录当前时间
//	/*kernek : calling function*/
//	matrix_mul_kernel << <numBlocks, threadsPerBlock >> > (dev_C, dev_A, dev_B, size, weight, height);
//	cudaDeviceSynchronize();
//	cudaError_t err = cudaGetLastError();
//	if (err != cudaSuccess) {
//		printf("CUDA ERROR: %s\n", cudaGetErrorString(err));
//	}
//	cudaEventRecord(stop, 0); //记录当前时间
//
//	cudaEventSynchronize(start); //Waits for an event to complete.
//	cudaEventSynchronize(stop);  //Waits for an event to complete.Record之前的任务
//	cudaEventElapsedTime(&time_elapsed, start, stop); //计算时间差
//	cout << "CUDA计算时间（不含数据传输）： " << time_elapsed << "ms" << endl;
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	/*Device To Host*/
//	cudaMemcpy(C, dev_C, size * sizeof(float), cudaMemcpyDeviceToHost);
//
//	/*Free Memory*/
//	cudaFree(dev_A);	cudaFree(dev_B);	cudaFree(dev_C);
//
//}
//
//int main()
//{
//	constexpr int weight{ num }; constexpr int height{ weight };
//	constexpr int size{ weight * height };
//	float* A = new float[size] {};
//	float* B = new float[size] {};
//	float* C = new float[size] {};
//	srand((unsigned)time(NULL));
//	for (int i = 0; i < size; ++i) {
//		//A[i] = (rand() % 4) / ((float)1); // 用来测试绝对误差标准
//		//B[i] = (rand() % 4) / ((float)1);
//		A[i] = (rand() % 256) / ((float)255); // 用来测试通用误差标准
//		B[i] = (rand() % 256) / ((float)255);
//	}
//	//cout << "*****A********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << A[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//	//cout << "*****B********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << B[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//	/*串行矩阵乘法计算*/
//	ULONGLONG t1, t2;
//	t1 = GetTickCount64();
//	float* CC = new float[size] {};
//	// nvcc 编译 parallel for 反而慢了
//	// Runtime工程：2000*2000要75秒，1000*1000要2秒，数据大了反而慢了
//	// CPP工程：2000*2000要5秒，1000*1000要0.15秒
////#pragma omp parallel for  
//	for (int i = 0; i < height; ++i) { // 立方级别倍增
//		for (int j = 0; j < weight; ++j) {
//			for (int col = 0; col < weight; ++col) {
//				CC[i * weight + j] += A[i * weight + col] * B[col * weight + j];
//			}
//		}
//	}// 3000*3000的串行需要时间 318s
//	t2 = GetTickCount64();
//	cout << "串行矩阵乘法计算耗时： " << t2 - t1 << "ms" << endl;
//
//	/*利用CUDA进行矩阵乘法计算*/
//	matrix_mul_withCuda(C, A, B, size, weight, height);
//
//	double error{ 0 };
//	for (int i = 0; i < height; ++i) {
//		for (int j = 0; j < weight; ++j) {
//			error += abs(CC[i * weight + j] - C[i * weight + j]);
//		}
//	}
//	cout << "sum of absolute error: " << error << endl;
//	cout << "mean error: " << error / size << endl;
//	//cout << "*****CC********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << CC[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//	//cout << "*****C********" << endl;
//	//for (int i = 0; i < size; ++i) {
//	//	cout << C[i] << "\t";
//	//	if ((i + 2) % weight == 1) cout << endl;
//	//}
//
//	return 0;
//}
///*调试此程序感悟：自己写的很多程序逻辑上可以简化从而减少代码量，这样不仅方便阅读也方便debug*/
///*逻辑性内容的调试程序手段： 有规律改变赋值、禁用部分逻辑等，观察预期和想的是否一样*/
///*我发现我的bug基本都不是逻辑bug，几乎都是打错字或者不小心没留意肌肉记忆顺写弄成的错误！关键是这些刻在自己身体上，还不好改！！*/
